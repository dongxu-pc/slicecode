/*************************************************************************
	> File Name: 07cultime.cu
	> Author: dong xu
	> Mail: gwmxyd@163.com 
	> Created Time: 2016年04月08日 星期五 10时18分09秒
 ************************************************************************/

#include <stdio.h>
#include <hip/hip_runtime.h>

//__global__声明的函数，告诉编译器这段代码交由CPU调用，由GPU执行
__global__ void mul(int *dev_a,const int NUM)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int dis=blockDim.x * gridDim.x;
    while(idx<NUM)
    {
        dev_a[idx]=dev_a[idx]%23*dev_a[idx]*5%9;
        idx+=dis;
    }
}

int main(void)
{
    const int thread_pre_block = 64;    //每个block的线程数量
    const int block_pre_grid = 8;    //grid中的block数量
    const int NUM = 45056;

    //申请主机内存，并进行初始化
    int host_a[NUM];
    for(int i=0;i<NUM;i++)
        host_a[i]=i;

    //定义cudaError，默认为cudaSuccess(0)
    hipError_t err = hipSuccess;

    //申请GPU存储空间
    int *dev_a;
    err=hipMalloc((void **)&dev_a, sizeof(int)*NUM);
    if(err!=hipSuccess)
    {
        perror("the hipMalloc on GPU is failed");
        return 1;
    }

    //将要计算的数据使用cudaMemcpy传送到GPU
    hipMemcpy(dev_a,host_a,sizeof(host_a),hipMemcpyHostToDevice);

    dim3 threads = dim3(thread_pre_block);
    dim3 blocks  = dim3(block_pre_grid);

    //使用event计算时间
    float time_elapsed=0;
    hipEvent_t start,stop;
    hipEventCreate(&start);    //创建Event
    hipEventCreate(&stop);

    hipEventRecord( start,0);    //记录当前时间
    mul<<<blocks, threads, 0, 0>>>(dev_a,NUM);
    hipEventRecord( stop,0);    //记录当前时间

    hipEventSynchronize(start);    //Waits for an event to complete.
    hipEventSynchronize(stop);    //Waits for an event to complete.Record之前的任务
    hipEventElapsedTime(&time_elapsed,start,stop);    //计算时间差

    hipMemcpy(&host_a,dev_a,sizeof(host_a),hipMemcpyDeviceToHost);    //计算结果回传到CPU

    hipEventDestroy(start);    //destory the event
    hipEventDestroy(stop);
    hipFree(dev_a);//释放GPU内存
    printf("执行时间：%f(ms)\n",time_elapsed);
    return 0 ;
}

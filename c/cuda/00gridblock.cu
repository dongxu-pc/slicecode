/*************************************************************************
	> File Name: 00gridblock.cu
	> Author: dong xu
	> Mail: gwmxyd@163.com 
	> Created Time: 2016年03月30日 星期三 23时27分51秒
 ************************************************************************/

#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void showKernel()
{
	int gdimx=gridDim.x;
	int gdimy=gridDim.y;
	int gdimz=gridDim.z;
	int bdimx=blockDim.x;
	int bdimy=blockDim.y;
	int bdimz=blockDim.z;
	int bidxx=blockIdx.x;
	int bidxy=blockIdx.y;
	int bidxz=blockIdx.z;
	int tidxx=threadIdx.x;
	int tidxy=threadIdx.y;
	int tidxz=threadIdx.z;
	int wsize=warpSize;

	printf("gridDim(%d,%d,%d),blockDim(%d,%d,%d)\nblockIdx(%d,%d,%d),threadIdx(%d,%d,%d)\nwarpSize:%d\n",gdimx,gdimy,gdimz,bdimx,bdimy,bdimz,bidxx,bidxy,bidxz,tidxx,tidxy,tidxz,wsize);
}

__global__ void addKernel(int a,int b,int* c)
{
	*c = a+b;
}

int main()
{
	int c;
	int *dev_c;
	hipMalloc((void**)&dev_c,sizeof(int));
	showKernel<<<5,3>>>();
	addKernel<<<1,1>>>(2,4,dev_c);

	hipMemcpy(&c,dev_c,sizeof(int),hipMemcpyDeviceToHost);
	printf("c=%d\n",c);
	hipFree(dev_c);

	return 0;
}

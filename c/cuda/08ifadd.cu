#include "hip/hip_runtime.h"
/*************************************************************************
	> File Name: 08ifadd.cu
	> Author: dong xu
	> Mail: gwmxyd@163.com 
	> Created Time: 2016年04月08日 星期五 10时23分20秒
 ************************************************************************/

#include <stdio.h>
#include <hip/hip_runtime.h>
#include "my_cuapi.h"

__global__ void ifaddKernel(int* c,const int * a,const int* b)
{
	int j = threadIdx.x;
	int i = threadIdx.x+1;
	if(i > 4)
		c[j]=a[j]+b[j];
	else
		c[j]=a[j]+b[j];
}

int main()
{
	int a[7]={1,2,3,4,5,6,7};
	int b[7]={10,20,30,40,50,60,70};
	int c[7];

	hipError_t cudaStatus;

	int *dev_a;
	int *dev_b;
	int *dev_c;

	cudaStatus = hipSetDevice(0);
	if(cudaStatus != hipSuccess){
		printf("There is no cuda devices.\n");
		exit(0);
	}

	cudaStatus = hipMalloc((void**)&dev_a,7*sizeof(int));
	HANDLE_ERROR(cudaStatus);
	cudaStatus = hipMalloc((void**)&dev_b,7*sizeof(int));
	HANDLE_ERROR(cudaStatus);
	cudaStatus = hipMalloc((void**)&dev_c,7*sizeof(int));
	HANDLE_ERROR(cudaStatus);

	cudaStatus = hipMemcpy(dev_a,a,7*sizeof(int),hipMemcpyHostToDevice);
	HANDLE_ERROR(cudaStatus);
	cudaStatus = hipMemcpy(dev_b,b,7*sizeof(int),hipMemcpyHostToDevice);
	HANDLE_ERROR(cudaStatus);

	hipEvent_t start,stop;
	float elaspedtime;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start,0);
	ifaddKernel<<<1,7>>>(dev_c,dev_a,dev_b);
	hipEventRecord(stop,0);

	hipEventSynchronize(start);
	hipEventSynchronize(stop);

	hipEventElapsedTime(&elaspedtime,start,stop);

	cudaStatus = hipMemcpy(c,dev_c,7*sizeof(int),hipMemcpyDeviceToHost);
	HANDLE_ERROR(cudaStatus);

	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	printf("Execute time:%f (ms)\n",elaspedtime);

	printf("a:");
	showIntArray(a,7);
	printf("\nb:");
	showIntArray(b,7);
	printf("\nc:");
	showIntArray(c,7);
	printf("\n");

	return 0;
}

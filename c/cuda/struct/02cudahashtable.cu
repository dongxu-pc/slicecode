/*************************************************************************
	> File Name: 02cudahashtable.cu
	> Author: dong xu
	> Mail: gwmxyd@163.com 
	> Created Time: 2016年04月02日 星期四 21时24分04秒
 ************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <time.h>

#include "hip/hip_runtime.h"


struct Entry{
	unsigned int key;
	void* value;
	struct 	Entry* next;
};

struct Table{
	size_t count;
	struct Entry **entries;
	struct Entry *pool;
	struct Entry *firstFree;
};

void init_table(struct Table* table,int entries,int elements)
{
	if(table == NULL)
		table = (struct Table*)malloc(sizeof(struct Table));
	table->count = entries;
	table->entries = (struct Entry**)calloc(entries,sizeof(struct Entry*));
	table->pool = (struct Entry*)malloc(elements * sizeof(struct Entry));
	table->firstFree = table->pool;
}

void free_table(struct Table* table)
{
	free(table->entries);
	free(table->pool);
	//if(table != NULL)
	//	free(table);
}

size_t hash(const unsigned int key,const size_t count)
{
	return key%count;
}

void add_to_table(struct Table *table,const unsigned int key,void*value)
{
	if(table == NULL){
		printf("Error:table is null\n");
		return;
	}
	size_t hashValue = hash(key,table->count);

	struct Entry* location = table->firstFree++;
	location->key = key;
	location->value = value;

	location->next = table->entries[hashValue];
	table->entries[hashValue] = location;
}

#define SIZE 100*1024*1024
#define ELEMENTS (SIZE/sizeof(unsigned int))

void verify_table(const struct Table *table)
{
	int count = 0;
	size_t i = 0;
	struct Entry* current;

	for(i=0;i<table->count;i++){
		current = table->entries[i];
		while(current != NULL){
			++count;
			if(hash(current->key,table->count) != i)
				printf("%d hashed to %ld,but was located at %ld\n",current->key,hash(current->key,table->count),i);
			current = current->next;
		}
	}

	if(count != ELEMENTS)
		printf("Hash Error!\n");
	else
		printf("Hash Success!\n");
}

#define HASH_ENTRIES 1024

int main()
{
	unsigned int *buffer = (unsigned int*)malloc(SIZE*sizeof(unsigned int));

	int i;
	clock_t start,stop;
	start = clock();

	struct Table* table = (struct Table*)malloc(sizeof(struct Table));
	init_table(table,HASH_ENTRIES,ELEMENTS);

	for(i=0;i<ELEMENTS;i++){
		add_to_table(table,buffer[i],(void*)NULL);
	}

	stop = clock();
	float elapsedTime = (float)(stop-start)/(float)CLOCKS_PER_SEC*1000.0f;
	printf("Time to hash:%3.1f ms\n",elapsedTime);

	verify_table(table);

	free_table(table);
	free(buffer);

	if(table != NULL)
		free(table);

	return 0;
}

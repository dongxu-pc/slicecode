#include "hip/hip_runtime.h"
/*************************************************************************
	> File Name: 05_0304.cu
	> Author: dong xu
	> Mail: gwmxyd@163.com 
	> Created Time: 2016年03月30日 星期三 13时37分15秒
 ************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

const int gridSize=50,blockSize=1024;
hipError_t addWithCuda(int *sum);

__global__ void addKernel(int *sum,int *timestamp)
{
	int i = blockIdx.x*blockDim.x +threadIdx.x + 1;
	sum[i] = atomicAdd(timestamp,1);
}

int main()
{
	int sum = 1;
	hipError_t cudaStatus;
	int num = 0;
	hipDeviceProp_t prop;

	cudaStatus = hipGetDeviceCount(&num);
	for(int i = 0;i<num;i++)
	{
		hipGetDeviceProperties(&prop,i);
	}
	cudaStatus = addWithCuda(&sum);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	printf("Final sum=%d\n",sum);
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) 
	{
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}
	return 0;
}

hipError_t addWithCuda(int *timestamp)
{
	int *dev_sum;
	int *dev_timestamp = 0;
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	int sum[gridsize*blocksize];

	hipEvent_t start,stop;
	float elapsedtime;

	clock_t cust,cufin;
	clock_t st,fin;

	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");  
		goto Error; 
	}

	cudaStatus = hipMalloc((void**)&dev_timestamp, sizeof(int));
	cudaStatus = hipMalloc((void**)&dev_sum,sizeof(int)*gridSize*blockSize);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!"); 
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_timestamp,timestamp,sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed!"); 
		goto Error;
	}

	hipEventCreate(&start);
	hipEventCreate(&stop);

	printf("addKernel<<<%d,%d>>>(%d)\n",gridSize,blockSize,*sum);
	hipEventRecord(start,0);
	atomic_inc_Kernel<<<gridSize,blockSize>>>(dev_timestamp);
    cudaStatus = hipDeviceSynchronize();
	hipEventRecord(stop,0);

	hipEventSynchronize(start);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedtime,start,stop);
	printf("Cuda Execute time:%f (ms)\n",elapsedtime);
	cudaStatus = hipMemcpy(timestamp,dev_timestamp,sizeof(int),hipMemcpyDeviceToHost);
	cudaStatus = hipMemcpy(sum,dev_sum,sizeof(int)*gridSize*blockSize,hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess){
		fprintf(stderr, "hipMemcpy failed!"); 
		goto Error;
	};

	printf("host sum=%d\n",*timestamp);

	hipEventDestroy(start);
	cudaEventDestory(stop);

Error:
	hipFree(dev_sum);
	return cudaStatus;
}



/*************************************************************************
	> File Name: 07atomic_pref.cu
	> Author: dong xu
	> Mail: gwmxyd@163.com 
	> Created Time: 2016年05月24日 星期二 15时49分40秒
 ************************************************************************/

#include <stdio.h>
#include <hip/hip_runtime.h>

#define GRIDSIZE 20
#define BLOCKSIZE 32

__global__ void atomic_shared_kernel(int* res)
{
	__shared__ int seq;

	if(threadIdx.x == 0)
		seq = 1;
	__syncthreads();

	atomicAdd(&seq,1);
	__syncthreads();

	if(threadIdx.x == 0)
		res[blockIdx.x] = seq;
}

__global__ void atomic_global_kernel(int* res)
{
	atomicAdd(res,1);
}

__global__ void shared_kernel(int* res)
{
	__shared__ int seq;

	if(threadIdx.x == 0)
		seq = blockIdx.x;
	__syncthreads();

	seq++;
	__syncthreads();

	if(threadIdx.x == 0)
		res[blockIdx.x] = seq;
}

__global__ void global_kernel(volatile int* res)
{
	*res++;
}

int main()
{
	int i;

	int* dev_seq_shared;
	int* dev_seq_global;

	int* seq_global;
	int* seq_shared;

	hipEvent_t start,stop;
	float time;

	hipError_t cudaStatus;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	seq_shared = (int*)malloc(GRIDSIZE*sizeof(int));
	seq_global = (int*)malloc(sizeof(int));

	*seq_global = 2;

	cudaStatus = hipMalloc((void**)&dev_seq_shared,GRIDSIZE*sizeof(int));
	cudaStatus = hipMalloc((void**)&dev_seq_global,sizeof(int));
	
	cudaStatus = hipMemcpy(dev_seq_global,seq_global,sizeof(int),hipMemcpyHostToDevice);

	hipEventRecord(start,0);
	global_kernel<<<GRIDSIZE,BLOCKSIZE>>>(dev_seq_global);
	hipEventRecord(stop,0);

	hipEventSynchronize(stop);
	hipEventElapsedTime(&time,start,stop);
	cudaStatus = hipMemcpy(seq_global,dev_seq_global,sizeof(int),hipMemcpyDeviceToHost);

	printf("global_kernel:seq = %d,time : %fms.\n",*seq_global,time);

	hipEventRecord(start,0);
	atomic_global_kernel<<<GRIDSIZE,BLOCKSIZE>>>(dev_seq_global);
	hipEventRecord(stop,0);

	hipEventSynchronize(stop);
	hipEventElapsedTime(&time,start,stop);
	cudaStatus = hipMemcpy(seq_global,dev_seq_global,sizeof(int),hipMemcpyDeviceToHost);
	printf("atomic_global_kernel:seq = %d,time : %fms.\n",*seq_global,time);

	
	hipEventRecord(start,0);
	shared_kernel<<<GRIDSIZE,BLOCKSIZE>>>(dev_seq_shared);
	hipEventRecord(stop,0);

	hipEventSynchronize(stop);
	hipEventElapsedTime(&time,start,stop);
	cudaStatus = hipMemcpy(seq_shared,dev_seq_shared,GRIDSIZE*sizeof(int),hipMemcpyDeviceToHost);
	printf("shared_kernel time : %fms.\n",time);
	for(i = 0;i < GRIDSIZE;i++){
		printf("%d ",seq_shared[i]);
		if(i%10 == 9)
			printf("\n");
	}
	printf("\n");

	
	hipEventRecord(start,0);
	atomic_shared_kernel<<<GRIDSIZE,BLOCKSIZE>>>(dev_seq_shared);
	hipEventRecord(stop,0);

	hipEventSynchronize(stop);
	hipEventElapsedTime(&time,start,stop);
	cudaStatus = hipMemcpy(seq_shared,dev_seq_shared,sizeof(int),hipMemcpyDeviceToHost);
	printf("atomic_kernel time : %fms.\n",time);
	for(i = 0;i < GRIDSIZE;i++){
		printf("%d ",seq_shared[i]);
		if(i%10 == 9)
			printf("\n");
	}
	printf("\n");


	//cudaEventDestory(start);
	//cudaEventDestory(stop);

	return 0;
}

#include "hip/hip_runtime.h"
/*************************************************************************
	> File Name: 05_0304.cu
	> Author: dong xu
	> Mail: gwmxyd@163.com 
	> Created Time: 2016年03月30日 星期三 13时37分15秒
 ************************************************************************/

#include <stdio.h>
#include <hip/hip_runtime.h>

const int gridSize=7,blockSize=1;
hipError_t addWithCuda(int *sum);

__global__ void addKernel(int *sum)
{
	int i = blockIdx.x*blockDim.x +threadIdx.x;
	int j=0;
	int tsum = 1;
	//atomicAdd(sum,i);
	//tsum = *tsum + i;
	//atomicCAS(sum,tsum,*sum);
	for(j=0;j<i;j++)
		tsum *=10; 
	for(j=0;j<tsum;j++);
	atomicAdd(sum,i);
}

int main()
{
	int sum = 1;
	hipError_t cudaStatus;
	int num = 0;
	hipDeviceProp_t prop;

	cudaStatus = hipGetDeviceCount(&num);
	for(int i = 0;i<num;i++)
	{
		hipGetDeviceProperties(&prop,i);
	}
	cudaStatus = addWithCuda(&sum);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	printf("Final sum=%d\n",sum);
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) 
	{
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}
	return 0;
}

hipError_t addWithCuda(int *sum)
{
	int *dev_sum = 0;
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");  
		goto Error; 
	}

	cudaStatus = hipMalloc((void**)&dev_sum, sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!"); 
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_sum,sum,sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed!"); 
		goto Error;
	}

	printf("addKernel<<<%d,%d>>>(%d)\n",gridSize,blockSize,*sum);
	addKernel<<<gridSize,blockSize>>>(dev_sum);
    cudaStatus = hipDeviceSynchronize();
	cudaStatus = hipMemcpy(sum,dev_sum,sizeof(int),hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed!"); 
		goto Error;
	}

Error:
	hipFree(dev_sum);
	return cudaStatus;
}



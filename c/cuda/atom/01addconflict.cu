#include "hip/hip_runtime.h"
/*************************************************************************
	> File Name: 05_0304.cu
	> Author: dong xu
	> Mail: gwmxyd@163.com 
	> Created Time: 2016年03月30日 星期三 13时37分15秒
 ************************************************************************/

#include <stdio.h>
#include <hip/hip_runtime.h>

const int gridSize=1,blockSize=7;
hipError_t addWithCuda(int *sum);

__global__ void addKernel(int *sum)
{
	int i = blockIdx.x+1;
	*sum = *sum + i;
	printf("thread %d:sum=%d\n",i,*sum);
}

int main()
{
	int sum = 1;
	hipError_t cudaStatus;
	int num = 0;
	hipDeviceProp_t prop;

	cudaStatus = hipGetDeviceCount(&num);
	for(int i = 0;i<num;i++)
	{
		hipGetDeviceProperties(&prop,i);
	}
	cudaStatus = addWithCuda(&sum);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	printf("Final sum=%d\n",sum);
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) 
	{
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}
	return 0;
}

hipError_t addWithCuda(int *sum)
{
	int *dev_sum = 0;
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");  
		goto Error; 
	}

	cudaStatus = hipMalloc((void**)&dev_sum, sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!"); 
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_sum,sum,sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed!"); 
		goto Error;
	}

	printf("addKernel<<<%d,%d>>>(%d)\n",gridSize,blockSize,*sum);
	addKernel<<<gridSize,blockSize>>>(dev_sum);
    cudaStatus = hipDeviceSynchronize();
	cudaStatus = hipMemcpy(sum,dev_sum,sizeof(int),hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed!"); 
		goto Error;
	}

Error:
	hipFree(dev_sum);
	return cudaStatus;
}



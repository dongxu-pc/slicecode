#include "hip/hip_runtime.h"
/*************************************************************************
	> File Name: 05_0304.cu
	> Author: dong xu
	> Mail: gwmxyd@163.com 
	> Created Time: 2016年03月30日 星期三 13时37分15秒
 ************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

const int gridSize=50,blockSize=1024;
hipError_t addWithCuda(int *sum);

__global__ void addKernel(int *sum)
{
	int i = blockIdx.x*blockDim.x +threadIdx.x + 1;
	//int j=0;
	//int tsum = 1;
	atomicAdd(sum,i);
	//tsum = *tsum + i;
	//atomicCAS(sum,tsum,*sum);
	//for(j=0;j<i;j++)
	//	tsum *=10; 
	//for(j=0;j<tsum;j++);
	//atomicAdd(sum,i);
}

void add(int *sum,int n)
{
	while(n>0)
	{
		*sum+=n;
		n--;
	}
}

int main()
{
	int sum = 1;
	hipError_t cudaStatus;
	int num = 0;
	hipDeviceProp_t prop;

	cudaStatus = hipGetDeviceCount(&num);
	for(int i = 0;i<num;i++)
	{
		hipGetDeviceProperties(&prop,i);
	}
	cudaStatus = addWithCuda(&sum);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	printf("Final sum=%d\n",sum);
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) 
	{
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}
	return 0;
}

hipError_t addWithCuda(int *sum)
{
	int *dev_sum = 0;
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);

	hipEvent_t start,stop;
	float elapsedtime;

	clock_t cust,cufin;
	clock_t st,fin;

	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");  
		goto Error; 
	}

	cudaStatus = hipMalloc((void**)&dev_sum, sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!"); 
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_sum,sum,sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed!"); 
		goto Error;
	}

	hipEventCreate(&start);
	hipEventCreate(&stop);

	printf("addKernel<<<%d,%d>>>(%d)\n",gridSize,blockSize,*sum);
	hipEventRecord(start,0);
	cust = clock();
	addKernel<<<gridSize,blockSize>>>(dev_sum);
    cudaStatus = hipDeviceSynchronize();
	cufin = clock();
	hipEventRecord(stop,0);

	hipEventSynchronize(start);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedtime,start,stop);
	printf("Cuda Execute time:%f (ms)\n",elapsedtime);
	elapsedtime = (float)(cufin-cust)*100/CLOCKS_PER_SEC;
	printf("Ho cuda Execute time:%f (ms)\n",elapsedtime);
	cudaStatus = hipMemcpy(sum,dev_sum,sizeof(int),hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess){
		fprintf(stderr, "hipMemcpy failed!"); 
		goto Error;
	};

	printf("cuda sum=%d\n",*sum);
	*sum = 1;

	st = clock();
	add(sum,gridSize*blockSize);
	fin = clock();
	elapsedtime = (float)(fin-st)*100/CLOCKS_PER_SEC;
	printf("Host Execute time:%f (ms)\n",elapsedtime);
	printf("host sum=%d\n",*sum);

//	hipEventDestroy(start);
//	cudaEventDestory(stop);

Error:
	hipFree(dev_sum);
	return cudaStatus;
}



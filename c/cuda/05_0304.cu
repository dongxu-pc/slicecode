#include "hip/hip_runtime.h"
/*************************************************************************
	> File Name: 05_0304.cu
	> Author: dong xu
	> Mail: gwmxyd@163.com 
	> Created Time: 2016年03月30日 星期三 13时37分15秒
 ************************************************************************/

#include <stdio.h>
#include <hip/hip_runtime.h>

hipError_t addWithCuda(int *c, const int *a);

__global__ void addKernel(int *c, const int *a)
{
	int i = blockIdx.x;
	*c = *a + i;
	printf("thread %d:a=%d,c=%d\n",i,*a,*c);
}

int main()
{
	int a = 1;
	int c = 0;
	hipError_t cudaStatus;
	int num = 0;
	hipDeviceProp_t prop;

	cudaStatus = hipGetDeviceCount(&num);
	for(int i = 0;i<num;i++)
	{
		hipGetDeviceProperties(&prop,i);
	}
	cudaStatus = addWithCuda(&c, &a);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	printf("a=%d,c=%d\n",a,c);
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) 
	{
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}
	return 0;
}

hipError_t addWithCuda(int *c, const int *a)
{
	int *dev_a = 0;
	int *dev_c = 0;
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");  
		goto Error; 
	}

	cudaStatus = hipMalloc((void**)&dev_c, sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!"); 
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&dev_a, sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!"); 
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_a, a,sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed!"); 
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_c, c,sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed!"); 
		goto Error;
	}

	printf("addKernel<<<7,1>>>(%d,%d)\n",*a,*c);
	addKernel<<<7,1>>>(dev_c,dev_a);
 //   cudaStatus = hipDeviceSynchronize();
	cudaStatus = hipMemcpy(c, dev_c,sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed!"); 
		goto Error;
	}
	printf("The anwser:%d\n",*c);

	printf("addKernel<<<1,7>>>(%d,%d)\n",*a,*c);
	addKernel<<<1,7>>>(dev_c,dev_a);

    cudaStatus = hipDeviceSynchronize();
	cudaStatus = hipMemcpy(c, dev_c,sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed!"); 
		goto Error;
	}
	printf("The anwser:%d\n",*c);

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	return cudaStatus;

}


